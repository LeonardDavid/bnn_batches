#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "cuda_kernel.h"
#include "netW.hpp"
#include "utils.cuh"

using namespace std;

/* need to flatten at runtime:

    -layer_1_weight[3][3][1][64]-
    -layer_4_weight[3][3][64][1]-
    -layer_8_weight[2048][49]-
    -layer_10_weight[10][32]-

    TODO:
    now: flatten arrays like layer7: unsigned long long *layer_7_output = (unsigned long long *) layer_6_output;
        maybe flatten in cpp file instead of cuda file
    later: to increase performance, have them flat in the file from the beggining

    cuda steps:
    // flatten 3D -> 1D arrays

    // prepare for kernel call
    // declare storage on device

    // allocate GPU device buffers

    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device

    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes


    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipEventRecord(start);
    // compute result - kernel call

    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // copy result from device to host

    cudaCheckErrors("CUDA memcpy failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // free the memory

    cudaCheckErrors("hipFree fail");

    // checksum

    return milliseconds;
*/

// Layer 1 - Convolution (xyz) 

__global__ void layer1_conv_kernel(unsigned char *d_cuda_layer_0_output, float *d_layer_1_bias, signed char *d_cuda_layer_1_weight, float *d_cuda_layer_1_output){

    // https://github.com/ULHPC/tutorials/blob/devel/cuda/exercises/convolution/LoG_gpu_solution.cu

    int N = (28+1); // +1 obligatory necessary because of reasons!
    int kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  // blockIdx.x? or .y?
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N + ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < BATCH_SIZE){
            if(m < NR_NEURONS) {
                d_cuda_layer_1_output[index4D_cuda(b,h,w,m,28,28,64)] = d_layer_1_bias[m];
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 28) {
                for (int kW = 0; kW < kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 28) {
                        if(b < BATCH_SIZE){
                            for (int c = 0; c < 1; c++) {
                                if(m < NR_NEURONS) {
                                    d_cuda_layer_1_output[index4D_cuda(b,h,w,m,28,28,64)] += d_cuda_layer_1_weight[index4D_cuda(kH,kW,c,m,3,1,64)] * d_cuda_layer_0_output[index4D_cuda(b,iH,iW,c,28,28,1)];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
}

float layer1_conv_cuda(unsigned char * const x, float * cuda_layer_1_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    // initialize layer_0_output where x is the input image
    unsigned char (*layer_0_output)[BATCH_SIZE][28][1] = (unsigned char (*)[BATCH_SIZE][28][1]) x;

    // flatten 3D -> 1D arrays
    // flatten layer_1_weight
    signed char *cuda_layer_1_weight = (signed char *) layer_1_weight;

    // flatten layer_0_output
    unsigned char *cuda_layer_0_output = (unsigned char *) layer_0_output;
    
    // prepare for kernel call
    // declare storage on device
    unsigned char *d_cuda_layer_0_output; // storage on device for cuda_layer_0_output
    float *d_layer_1_bias; // storage on device for layer_1_bias
    signed char *d_cuda_layer_1_weight; // storage on device for cuda_layer_1_weight
    float *d_cuda_layer_1_output; // RESULT storage on device for cuda_layer_1_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_0_output, BATCH_SIZE*784*sizeof(unsigned char)); // 784 = 28x28 dim of cuda_layer_0_output
    hipMalloc((void **) &d_layer_1_bias, 64*sizeof(float)); // 64 = dim of layer_1_bias
    hipMalloc((void **) &d_cuda_layer_1_weight, 3*3*1*64*sizeof(signed char)); // 576 = 3x3x1x64 dim of layer_1_weight
    hipMalloc((void **) &d_cuda_layer_1_output, BATCH_SIZE*50176*sizeof(float)); // 50176 = 28x28x64 dim of layer_1_output
    cudaCheckErrors("Failed to allocate device buffer");

    // hipMemGetInfo(&free,&total);   
    // printf("after: %d KB free of total %d KB\n",free/1024,total/1024);

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_0_output, cuda_layer_0_output, (BATCH_SIZE*784*sizeof(unsigned char)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_1_bias, layer_1_bias, (64*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_1_weight, cuda_layer_1_weight, (3*3*1*64*sizeof(signed char)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 28;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 28;
    const int GRIDZSIZE = NR_NEURONS;
    
    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer1_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_0_output, d_layer_1_bias, d_cuda_layer_1_weight, d_cuda_layer_1_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_1_output, d_cuda_layer_1_output, (BATCH_SIZE*50176*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
    
    // free the memory
    hipFree(d_cuda_layer_0_output);
    hipFree(d_layer_1_bias);
    hipFree(d_cuda_layer_1_weight);
    hipFree(d_cuda_layer_1_output);
    cudaCheckErrors("hipFree fail");
    
    // // checksum L1 = -605468.812500
    // float sum = 0;
    // ofstream g("layer_1_par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*50176;i<(b+1)*50176;i++){
    //         sum += cuda_layer_1_output[i];
    //         g<<cuda_layer_1_output[i]<<" ";  
    //         if((i+1)%64==0){
    //             g<<endl;
    //         }
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}

// Layer 2 - Maxpool (xyz)

__global__ void layer2_maxpool_kernel(float *d_cuda_layer_1_output, float *d_cuda_layer_2_output, float lowest){

    int N = (14+1); // +1 obligatory necessary because of reasons!
    int kernel_size = 2;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int c = blockIdx.z; // neurons in z-dir

    int b = blockIdx.x; // Batches index in grid x dir
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b<BATCH_SIZE){
            if(c<NR_NEURONS) {
                d_cuda_layer_2_output[index4D_cuda(b,h,w,c,14,14,64)] = lowest;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH<kernel_size; kH++){
            for (int kW = 0; kW<kernel_size; kW++){
                if(b<BATCH_SIZE){
                    if(c<NR_NEURONS) {
                        d_cuda_layer_2_output[index4D_cuda(b,h,w,c,14,14,64)] = fmax(d_cuda_layer_1_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,28,28,64)], d_cuda_layer_2_output[index4D_cuda(b,h,w,c,14,14,64)]);
                    }
                }
            }
        }
    }
}

float layer2_maxpool_cuda(float * cuda_layer_1_output, float * cuda_layer_2_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // no 3D arrays to be flattened

    // prepare for kernel call
    // declare storage on device
    float *d_cuda_layer_1_output; // storage on device for cuda_layer_1_output
    float *d_cuda_layer_2_output; // RESULT storage on device for cuda_layer_2_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_1_output, BATCH_SIZE*50176*sizeof(float)); // 50176 = 28x28x64 dim of layer_1_output
    hipMalloc((void **) &d_cuda_layer_2_output, BATCH_SIZE*12544*sizeof(float)); // 12544 = 14x14x64 dim of layer_2_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_1_output, cuda_layer_1_output, (BATCH_SIZE*50176*sizeof(float)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 14;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 14;
    const int GRIDZSIZE = NR_NEURONS;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // std library not allowed on device
    const float LOWEST = std::numeric_limits<float>::lowest();

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer2_maxpool_kernel<<<numBlocks, threadsPerBlock>>>(d_cuda_layer_1_output, d_cuda_layer_2_output, LOWEST);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_2_output, d_cuda_layer_2_output, (BATCH_SIZE*12544*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_1_output);
    hipFree(d_cuda_layer_2_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L2 = 455610.125000
    // float sum = 0;
    // ofstream g("layer_2_par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*12544;i<(b+1)*12544;i++){
    //         sum += cuda_layer_2_output[i];
    //         g<<cuda_layer_2_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}

// Layer 4 - Convolution (xyz)

__global__ void layer4_conv_kernel(unsigned long long *d_cuda_layer_3_output, float *d_layer_4_bias, unsigned long long *d_cuda_layer_4_weight, signed short *d_cuda_layer_4_output){
    
    int N = (14+1); // +1 obligatory necessary because of reasons!
    int kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    int b = blockIdx.x; //batches in x-dir
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b<BATCH_SIZE){
            if(m<NR_NEURONS) {
                d_cuda_layer_4_output[index4D_cuda(b,h,w,m,14,14,64)] = d_layer_4_bias[m]; // = 0;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH<kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 14) {
                for (int kW = 0; kW<kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 14) {
                        if(b<BATCH_SIZE){
                            if(m<NR_NEURONS) {
                                for (int c = 0; c < 1; c++) {
                                    d_cuda_layer_4_output[index4D_cuda(b,h,w,m,14,14,64)] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_4_weight[index4D_cuda(kH,kW,m,c,3,64,1)] ^ d_cuda_layer_3_output[index4D_cuda(b,iH,iW,c,14,14,64)])) - 64;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

float layer4_conv_cuda(unsigned long long * cuda_layer_3_output, signed short * cuda_layer_4_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_4_weight
    unsigned long long *cuda_layer_4_weight = (unsigned long long *) layer_4_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_3_output; // storage on device for cuda_layer_3_output
    float *d_layer_4_bias; // storage on device for layer_4_bias
    unsigned long long *d_cuda_layer_4_weight; // storage on device for cuda_layer_4_weight
    signed short *d_cuda_layer_4_output; // RESULT storage on device for cuda_layer_4_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_3_output, BATCH_SIZE*12544*sizeof(unsigned long long)); // 196=14x14 dim of cuda_layer_4_output
    hipMalloc((void **) &d_layer_4_bias, 64*sizeof(float)); // 64 = dim of layer_4_bias
    hipMalloc((void **) &d_cuda_layer_4_weight, 3*3*64*1*sizeof(unsigned long long)); // 576 = 3x3x64x[1x64] dim of layer_4_weight [ULL]
    hipMalloc((void **) &d_cuda_layer_4_output, BATCH_SIZE*12544*sizeof(signed short)); // 12544 = 14x14x64 dim of layer_4_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_3_output, cuda_layer_3_output, (BATCH_SIZE*12544*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_4_bias, layer_4_bias, (64*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_4_weight, cuda_layer_4_weight, (3*3*64*1*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 14;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 14;
    const int GRIDZSIZE = NR_NEURONS;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer4_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_3_output, d_layer_4_bias, d_cuda_layer_4_weight, d_cuda_layer_4_output);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");    
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_4_output, d_cuda_layer_4_output, (BATCH_SIZE*12544*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_3_output);
    hipFree(d_layer_4_bias);
    hipFree(d_cuda_layer_4_weight);
    hipFree(d_cuda_layer_4_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L4 = 6334.000000
    // float sum = 0;
    // ofstream g("layer_4_par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*12544;i<(b+1)*12544;i++){
    //         sum += cuda_layer_4_output[i];
    //         g<<cuda_layer_4_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}

// Layer 5 - Maxpool (xyz)
__global__ void layer5_maxpool_kernel(signed short * d_cuda_layer_4_output, signed short * d_cuda_layer_5_output, signed short lowest){

    int N = (7+1); // +1 obligatory necessary because of reasons!
    int kernel_size = 2;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int c = blockIdx.z; // neurons in z-dir

    int b = blockIdx.x; // batches in x-dir
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b<BATCH_SIZE){
            if(c<NR_NEURONS) {
                d_cuda_layer_5_output[index4D_cuda(b,h,w,c,7,7,64)] = lowest;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH<kernel_size; kH++){
            for (int kW = 0; kW<kernel_size; kW++){
                if(b<BATCH_SIZE){
                    if(c<NR_NEURONS) {
                        d_cuda_layer_5_output[index4D_cuda(b,h,w,c,7,7,64)] = 
                        (d_cuda_layer_4_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,14,14,64)] >= d_cuda_layer_5_output[index4D_cuda(b,h,w,c,7,7,64)]) ? 
                        d_cuda_layer_4_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,14,14,64)] : d_cuda_layer_5_output[index4D_cuda(b,h,w,c,7,7,64)];
                    }
                }
            }
        }
    }
}

float layer5_maxpool_cuda(signed short * cuda_layer_4_output, signed short * cuda_layer_5_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // no arrays to be flattened

    // prepare for kernel call
    // declare storage on device
    signed short *d_cuda_layer_4_output; // storage on device for cuda_layer_4_output
    signed short *d_cuda_layer_5_output; // RESULT storage on device for cuda_layer_5_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_4_output, BATCH_SIZE*12544*sizeof(signed short)); // 12544 = 14x14xx64 dim of layer_4_output
    hipMalloc((void **) &d_cuda_layer_5_output, BATCH_SIZE*3136*sizeof(signed short)); // 3136 = 7x7x64 dim of layer_5_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_4_output, cuda_layer_4_output, (BATCH_SIZE*12544*sizeof(signed short)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 7;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 7;
    const int GRIDZSIZE = NR_NEURONS;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // std library not allowed on device
    const signed short LOWEST = std::numeric_limits<signed short>::lowest();

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer5_maxpool_kernel<<<numBlocks, threadsPerBlock>>>(d_cuda_layer_4_output, d_cuda_layer_5_output, LOWEST);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_5_output, d_cuda_layer_5_output, (BATCH_SIZE*3136*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_4_output);
    hipFree(d_cuda_layer_5_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L5 = 81406.0
    // float sum = 0;
    // ofstream g("layer_5_par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*3136;i<(b+1)*3136;i++){
    //         sum += cuda_layer_5_output[i];
    //         g<<cuda_layer_5_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}

// Layer 6 - Step
// skipped for now

// Layer 8 - Gemm (xyz)
__global__ void layer8_gemm_kernel(unsigned long long *d_cuda_layer_7_output, float *d_layer_8_bias, unsigned long long *d_cuda_layer_8_weight, signed short *d_cuda_layer_8_output){

    int z = blockDim.x * blockIdx.z + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int d = z*blockDim.x+y;

    int b = blockIdx.x;

    if(d < 2048){
        if(b < BATCH_SIZE){
            d_cuda_layer_8_output[b*2048 + d] = d_layer_8_bias[d];
            for (int i = 0; i < 49; i++) {
                d_cuda_layer_8_output[b*2048 + d] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_8_weight[d*49+i] ^ d_cuda_layer_7_output[b*49+i])) - 64;
            }
        }
    }
}

float layer8_gemm_cuda(unsigned long long * cuda_layer_7_output, signed short * cuda_layer_8_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_8_weight
    unsigned long long *cuda_layer_8_weight = (unsigned long long *) layer_8_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_7_output; // storage on device for cuda_layer_7_output
    float *d_layer_8_bias;  // storage on device for layer_8_bias
    unsigned long long *d_cuda_layer_8_weight; // storage on device for cuda_layer_8_weight
    signed short *d_cuda_layer_8_output; // RESULT storage on device for cuda_layer_8_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_7_output, BATCH_SIZE*49*sizeof(unsigned long long)); // 49=7x7 dim of cuda_layer_7_output
    hipMalloc((void **) &d_layer_8_bias, 2048*sizeof(float)); // 2048 = dim of layer_8_bias
    hipMalloc((void **) &d_cuda_layer_8_weight, 2048*49*sizeof(unsigned long long)); // 100352 = 2048x49 dim of layer_8_weight [ULL]
    hipMalloc((void **) &d_cuda_layer_8_output, BATCH_SIZE*2048*sizeof(signed short)); // 2048 = dim of layer_8_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_7_output, cuda_layer_7_output, (BATCH_SIZE*49*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_8_bias, layer_8_bias, (2048*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_8_weight, cuda_layer_8_weight, (2048*49*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    /*
        Maximum threads in a block: 1024 => Maximum block size 32x32
        if more than 1024 threads are needed, then set block size to maximum (32x32) and put multiple blocks in z-dir
        else if less than 1024 are needed, then only create 1 (square) block in z-dir, of size ceil(sqrt(THREADS_NEEDED))
    */
    const int BLKXSIZE = 32;
    const int BLKYSIZE = 32;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 1;
    const int GRIDZSIZE = 2;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer8_gemm_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_7_output, d_layer_8_bias, d_cuda_layer_8_weight, d_cuda_layer_8_output);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_8_output, d_cuda_layer_8_output, (BATCH_SIZE*2048*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_7_output);
    hipFree(d_layer_8_bias);
    hipFree(d_cuda_layer_8_weight);
    hipFree(d_cuda_layer_8_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L8 = 8936.000000
    // float sum = 0;
    // ofstream g("layer_8_par1.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*2048;i<(b+1)*2048;i++){
    //         sum += cuda_layer_8_output[i];
    //         g<<cuda_layer_8_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}

// Layer 10 - Gemm (xyz)
__global__ void layer10_gemm_kernel(unsigned long long *d_cuda_layer_9_output, float *d_layer_10_bias, unsigned long long *d_cuda_layer_10_weight, signed short *d_cuda_layer_10_output){

    int z = blockDim.x * blockIdx.z + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int d = z*blockDim.x+y;

    int b = blockIdx.x;

    if(d < 10){
        if(b<BATCH_SIZE){
            d_cuda_layer_10_output[b*10 + d] = d_layer_10_bias[d];
            for (int i = 0; i < 32; i++) {
                d_cuda_layer_10_output[b*10 + d] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_10_weight[d*32+i] ^ d_cuda_layer_9_output[b*32+i])) - 64;
            }
        }
    }
}

float layer10_gemm_cuda(unsigned long long * cuda_layer_9_output, signed short * cuda_layer_10_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_10_weight
    unsigned long long *cuda_layer_10_weight = (unsigned long long *) layer_10_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_9_output; // storage on device for cuda_layer_9_output
    float *d_layer_10_bias;  // storage on device for layer_10_bias
    unsigned long long *d_cuda_layer_10_weight; // storage on device for cuda_layer_10_weight
    signed short *d_cuda_layer_10_output; // RESULT storage on device for cuda_layer_10_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_9_output, BATCH_SIZE*32*sizeof(unsigned long long)); // 32 = dim of cuda_layer_9_output
    hipMalloc((void **) &d_layer_10_bias, 10*sizeof(float)); // 10 = dim of layer_10_bias
    hipMalloc((void **) &d_cuda_layer_10_weight, 10*32*sizeof(unsigned long long)); // 320 = 32x10 dim of layer_10_weight [ULL]
    hipMalloc((void **) &d_cuda_layer_10_output, BATCH_SIZE*10*sizeof(signed short)); // 10 = dim of layer_10_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_9_output, cuda_layer_9_output, (BATCH_SIZE*32*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_10_bias, layer_10_bias, (10*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_10_weight, cuda_layer_10_weight, (10*32*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    /*
        Maximum threads in a block: 1024 => Maximum block size 32x32
        if more than 1024 threads are needed, then set block size to maximum (32x32) and put multiple blocks in z-dir
        else if less than 1024 are needed, then only create 1 (square) block in z-dir, of size ceil(sqrt(THREADS_NEEDED))
    */
    const int BLKXSIZE = 4;
    const int BLKYSIZE = 4;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 1;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer10_gemm_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_9_output, d_layer_10_bias, d_cuda_layer_10_weight, d_cuda_layer_10_output);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // copy result from device to host
    hipMemcpy(cuda_layer_10_output, d_cuda_layer_10_output, (BATCH_SIZE*10*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_9_output);
    hipFree(d_layer_10_bias);
    hipFree(d_cuda_layer_10_weight);
    hipFree(d_cuda_layer_10_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L10 = -666.000000
    // float sum = 0;
    // ofstream g("layer_10_par1.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*10;i<(b+1)*10;i++){
    //         sum += cuda_layer_10_output[i];
    //         g<<cuda_layer_10_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    // cout<<endl;
    return milliseconds;
}
